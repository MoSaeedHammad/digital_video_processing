
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>     
#include <fstream>
#include <chrono>

#define IMAGE_WIDTH 3840
#define IMAGE_HEIGH 2160
#define IMAGE_CH 3u
#define IMAGE_OFF 54u


__global__
void kernelYUV2RGB(unsigned char *a, unsigned char *b) {
    int i = 3*blockIdx.x;
	int c = b[i+0] - 16;
    int d = b[i+1] - 128;
    int e = b[i+2] - 128;
    a[i+0]=( 298 * c + 409 * e + 128) >> 8;
    a[i+1]= ( 298 * c - 100 * d - 208*e + 128) >> 8;
    a[i+2] = ( 298 * c + 516 * d + 128) >> 8;
}

__global__
void kernelRGB2YUV(unsigned char *a, unsigned char *db) {
	int i = 3*blockIdx.x;
	int r = db[i+0];
    int g = db[i+1];
    int b = db[i+2];
	a[i+0] = ((66*r + 129*g +25*b + 128) >> 8 ) + 16 ;
	a[i+1] = ((-38*r - 74*g +112*b + 128) >> 8 ) + 128 ;
	a[i+2] = ((112*r - 94*g  -18*b + 128) >> 8 ) + 128 ;
}

__global__
void kernelRGB2YUV422(unsigned char *a, unsigned char *db) {
	int i = 3*blockIdx.x;
	int r = db[i+0];
    int g = db[i+1];
    int b = db[i+2];
	int r1 = db[i+0+3];
    int g1 = db[i+1+3];
    int b1 = db[i+2+3];
	int u1,u2;
	a[i+0] = ((66*r + 129*g +25*b + 128) >> 8 ) + 16 ;
	a[i+0+3] = ((66* r1 + 129*g1 +25* b1 + 128) >> 8 ) + 16 ;
	u1 = ((-38*r - 74*g +112*b + 128) >> 8 ) + 128 ;
	u2 = ((-38*r1 - 74* g1 +112*b1 + 128) >> 8 ) + 128 ;
	a[i+1] = a[i+1+3] = (u1+u2)/2;
	u1 = ((112*r - 94*g  -18*b + 128) >> 8 ) + 128 ;
	u2 = ((112*r1 - 94* g1  -18*b1 + 128) >> 8 ) + 128 ;
	a[i+2] = a[i+2+3] = (u1+u2)/2;	
}
using namespace std;
using namespace std::chrono;

double calculatePSNR(unsigned char* bufYUV, unsigned char* bufRGB, unsigned int len)
{
    double MSER = 0;
    double MSEG = 0;
    double MSEB = 0;
    double MSE = 0;
    double PSNR =0;

    for(unsigned int i = 0 ; i < len;i++)
    {
        MSE += pow((bufYUV[i] -bufRGB[i]),2);
    }
    MSE = (MSE)/(len);
    PSNR = 10*log10((255.0*255.0)/MSE);
    std::cout<<"MSE "<<MSE<<std::endl;
    return PSNR;
}

bool readImageSimd(std::string imageName ,  char* buf )
{
    bool retVal = true ;
    int off = 0;
    int wdth = 0 ;
    int height = 0;
    std::ifstream ImageFile(imageName, std::ios::binary | std::ios::ate);
    if(ImageFile.fail())
        return false;
    //temproray buffer to hold the image as char buffer
    auto sz= ImageFile.tellg();
    //temproray buffer to hold the image as char buffer
    std::cout<<"reading "<<sz<<std::endl;
    ImageFile.seekg(0, std::ios::beg);
    ImageFile.read(buf, sz);
    ImageFile.close();
    return  true;
}
void converRGBtoYUV(unsigned char* bufRGB,unsigned char *bufYUV,unsigned int len)
{
    int off = IMAGE_OFF;
    for(int i = 0 ; i < off ;i++)
        bufYUV[i] = bufRGB[i];
    //for each pixel
    for(int i = off ; i < len ; i+=3)
    {
        int r = bufRGB[i+0];
        int g = bufRGB[i+1];
        int b = bufRGB[i+2];

        bufYUV[i+0] = ((66*r + 129*g +25*b + 128) >> 8 ) + 16 ;
        bufYUV[i+1] = ((-38*r - 74*g +112*b + 128) >> 8 ) + 128 ;
        bufYUV[i+2] = ((112*r - 94*g  -18*b + 128) >> 8 ) + 128 ;
    }
}
static unsigned char SIMD_bufRGB[IMAGE_WIDTH*IMAGE_HEIGH*IMAGE_CH + IMAGE_OFF];
static unsigned char SIMD_bufRGB1[IMAGE_WIDTH*IMAGE_HEIGH*IMAGE_CH + IMAGE_OFF];
static unsigned char SIMD_bufYUV[IMAGE_WIDTH*IMAGE_HEIGH*IMAGE_CH + IMAGE_OFF];

int main() {
	string fileName1 = "testo.bmp";

	unsigned int  len = (IMAGE_WIDTH*IMAGE_HEIGH*IMAGE_CH + IMAGE_OFF);
	unsigned char *da, *db;
	hipMalloc((void **)&da, len*sizeof(char));
    hipMalloc((void **)&db, len*sizeof(char));

	
    cout<<"[+]Testing using SIMD\n";
    //read Image from buffer directly so data are next to each other
    readImageSimd(fileName1,(char*) SIMD_bufRGB);
    cout<<"[+]Converting to YUV\n";
	auto start = high_resolution_clock::now();
	hipMemcpy(db, SIMD_bufRGB, len*sizeof(char), hipMemcpyHostToDevice);   
    kernelRGB2YUV<<<IMAGE_WIDTH*IMAGE_HEIGH, 1>>>(da, db);
	hipMemcpy(SIMD_bufYUV, da, len*sizeof(char), hipMemcpyDeviceToHost);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);
    cout<<"[+]Elapsed "<<duration.count() << " ms" <<endl;
    //converRGBtoYUV(SIMD_bufRGB,SIMD_bufYUV,len);
	
	cout<<"[+]Converting to YUV422\n";
	start = high_resolution_clock::now();
	hipMemcpy(db, SIMD_bufRGB, len*sizeof(char), hipMemcpyHostToDevice);   
    kernelRGB2YUV422<<<(IMAGE_WIDTH*IMAGE_HEIGH)/2, 1>>>(da, db);
	hipMemcpy(SIMD_bufYUV, da, len*sizeof(char), hipMemcpyDeviceToHost);
    stop = high_resolution_clock::now();
    duration = duration_cast<milliseconds>(stop - start);
    
	cout<<"[+]Converting to RGB\n";
	start = high_resolution_clock::now();
	hipMemcpy(db, SIMD_bufYUV, len*sizeof(char), hipMemcpyHostToDevice);   
    kernelYUV2RGB<<<IMAGE_WIDTH*IMAGE_HEIGH, 1>>>(da, db);
	hipMemcpy(SIMD_bufRGB1, da, len*sizeof(char), hipMemcpyDeviceToHost);
    stop = high_resolution_clock::now();
    duration = duration_cast<milliseconds>(stop - start);
    cout<<"[+]Elapsed "<<duration.count() << " ms" <<endl;
    hipFree(da);
    hipFree(db);
	for(int i = 0 ; i < IMAGE_OFF ; i++)
		SIMD_bufRGB1[i] = SIMD_bufYUV[i];

	cout<<"[+]Calculating PSNR\n";
    double psnr = calculatePSNR((unsigned char*)SIMD_bufRGB1,(unsigned char*)SIMD_bufRGB,len);
    cout<<"psnr "<<psnr<<endl;

    return 0;
}